#include "hip/hip_runtime.h"
#include <>
#include "ReconstructorCUDA.cuh"
#include "fileReaderCUDA.cuh"
namespace SLS
{

ReconstructorCUDA :: ReconstructorCUDA(const size_t projX, const size_t projY): 
    Reconstructor()
{
    projector_ = new Projector(projX, projY);
}
ReconstructorCUDA::~ReconstructorCUDA(){
    for (auto &cam: cameras_)
        delete cam;
    delete projector_;
}
void ReconstructorCUDA::addCamera(Camera *cam)
{
    cameras_.push_back(cam);
}
void ReconstructorCUDA::reconstruct()
{
    // For each camera, hack
    GPUBuckets buckets[2] =
    {
        GPUBuckets( 1024*768,110),
        GPUBuckets( 1024*768,110)
    };
    
    /**** Profile *****/
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    /**/


    for(size_t camIdx = 0; camIdx < cameras_.size(); camIdx++)
    {
        FileReaderCUDA* cam = (FileReaderCUDA*)cameras_[camIdx];
        LOG::writeLog("Generating reconstruction bucket for \"%s\" ... \n", cam->getName().c_str());
        cam->computeShadowsAndThresholds();    // can it be done in GPU? Yes
        size_t x=0,y=0,xTimesY=0;
        cam->getResolution(x,y);
        xTimesY=x*y;
        //Skip first two frames
        cam->getNextFrame();
        cam->getNextFrame();
        // Load all images into GPU memory
        uchar *images_d=nullptr;
        gpuErrchk(hipMalloc((void**)&images_d, sizeof(uchar)*xTimesY*projector_->getRequiredNumFrames()*2));
        Dynamic_Bitset_Array bitsetArray(xTimesY, projector_->getRequiredNumFrames());

        //Preparing data
        for (size_t i=0; i<projector_->getRequiredNumFrames(); i++)
        {
            auto frm = cam->getNextFrame();
            auto invFrm = cam->getNextFrame();
            assert(frm.isContinuous() && invFrm.isContinuous());
            gpuErrchk( hipMemcpy( &images_d[xTimesY*2*i], frm.data, 
                    sizeof(uchar)*xTimesY, hipMemcpyHostToDevice));
            gpuErrchk( hipMemcpy( &images_d[xTimesY*(2*i+1)], invFrm.data, 
                    sizeof(uchar)*xTimesY, hipMemcpyHostToDevice));
        }

        Kernel::genPatternArray<<<200,200>>> (
                images_d, 
                projector_->getRequiredNumFrames(),
                xTimesY,
                cam->getWhiteThreshold(0),
                cam->getMask()->getGPUOBJ(),
                bitsetArray.getGPUOBJ()
                );
        //Check for errors
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipFree(images_d)); // Release the heavy image array

        Kernel::buildBuckets<<<200, 200>>> (
             cam->getMask()->getGPUOBJ(),
             bitsetArray.getGPUOBJ(),
             xTimesY,
             buckets[camIdx].getGPUOBJ()
            );
        gpuErrchk(hipPeekAtLastError());
    }


    // A lot of hacks down there, need to be refactored
    auto camera0 = (FileReaderCUDA*)(cameras_[0]);
    auto camera1 = (FileReaderCUDA*)(cameras_[1]);
    float* cloud = nullptr;
    size_t resX, resY;
    camera0->getResolution(resX, resY);

    gpuErrchk ( hipMalloc((void**)&cloud, buckets[0].getNumBKTs()*sizeof(float)*4));

    // Reconstructing point cloud
    LOG::writeLog("Reconstructing point cloud ...\n");
    Kernel::getPointCloud2Cam<<<200,200>>>(
            buckets[0].getGPUOBJ(),
            camera0->getMask()->getGPUOBJ(),
            camera0->getDeviceCamMat(),
            camera0->getDeviceDistMat(),
            camera0->getDeviceCamTransMat(),

            buckets[1].getGPUOBJ(),
            camera1->getMask()->getGPUOBJ(),
            camera1->getDeviceCamMat(),
            camera1->getDeviceDistMat(),
            camera1->getDeviceCamTransMat(),
            resX,resY,
            cloud
            );
    gpuErrchk( hipPeekAtLastError());
    pointCloud_.resize(buckets[0].getNumBKTs()*4);
    gpuErrchk( 
            hipMemcpy(  &pointCloud_[0],cloud, buckets[0].getNumBKTs()*sizeof(float)*4, hipMemcpyDeviceToHost));
    /**** Profile *****/
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    LOG::writeLog("GPU Time : %fms\n", milliseconds);
    /*****/
    gpuErrchk( hipFree(cloud));
    LOG::writeLog("Done\n");
}

namespace Kernel{
// Kernels 

__global__ void genPatternArray(
        const uchar * imgs,
        size_t numImgs,
        size_t XtimesY,
        uchar whiteThreshold,
        Dynamic_Bitset_Array_GPU mask,
        Dynamic_Bitset_Array_GPU patterns
        )
{
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < XtimesY)   // For each pixel
    {
        for (size_t i = 0; i<numImgs; i++)
        {
            if (!mask.getBit(0, idx)) 
            {
                // set the bit to black, unnecessary
                for (size_t j=0; j<numImgs; j++)
                    patterns.clearBit(j, idx);
                continue;
            }
            uchar pixel = imgs[ idx + XtimesY*(2*i)];
            uchar invPixel = imgs[ idx + XtimesY*(2*i+1)];
            if (invPixel > pixel && invPixel-pixel >= whiteThreshold)
                patterns.clearBit(numImgs-1-i, idx);
            else if (pixel > invPixel && pixel-invPixel > whiteThreshold)
                patterns.setBit(numImgs-1-i, idx);
            else
                mask.clearBit(0, idx);
        }
        idx += stride;
    }
}


__global__ void buildBuckets(
        Dynamic_Bitset_Array_GPU mask,
        Dynamic_Bitset_Array_GPU patterns,
        size_t XtimesY,

        GPUBucketsObj bkts
        )
{
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < XtimesY)   // For each pixel
    {
        glm::uvec2 bkt2v = patterns.to_uint_gray(idx);
        if (bkt2v.x < 1024 && bkt2v.y < 768 && mask.getBit(0, idx))
            bkts.add2Bucket(idx, bkt2v.x+bkt2v.y*1024);
        idx += stride;
    }
}

__global__ void getPointCloud2Cam(
        GPUBucketsObj buckets0,
        Dynamic_Bitset_Array_GPU mask0,
        float *camMat0,
        float *distMat0,
        float *camTransMat0,

        GPUBucketsObj buckets1,
        Dynamic_Bitset_Array_GPU mask1,
        float *camMat1,
        float *distMat1,
        float *camTransMat1,


        uint camResX,
        uint camResY,

        float* pointCloud
        )
{
    //Each thread takes care of one projector pixel
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < buckets0.NUM_BKTS_)   // For each pixel
    {
        if ( buckets0.count_[idx] == 0 || buckets1.count_[idx] == 0) 
        {
            memset( &pointCloud[4*idx], 0, sizeof(float)*4);
            pointCloud[4*idx+3] = float(0.0);
        }
        else
        {
            //Undistorted pixels
            float minDist = 99999.0;
            float minMidPoint[4];

            float avgPoint[4];
            memset(avgPoint, 0, sizeof(float)*4);
            uint ptCount = 0;

            for (uint i=0; i<buckets0.count_[idx]; i++)
                for (uint j=0; j<buckets1.count_[idx]; j++)
                {

                    float undistorted0[2];
                    float undistorted1[2];

                    //Pick the first pixel in both buckets to test
                    undistortPixel(
                            buckets0.data_[idx*buckets0.MAX_CNT_PER_BKT_+i],
                            camResX, camResY,
                            camMat0, distMat0,
                            undistorted0);
                    undistortPixel(
                            buckets1.data_[idx*buckets1.MAX_CNT_PER_BKT_+j],
                            camResX, camResY,
                            camMat1, distMat1,
                            undistorted1);

                    float origin0[4];
                    float origin1[4];
                    float dir0[4];
                    float dir1[4];

                    getRay(undistorted0, camMat0, camTransMat0, 
                            origin0, dir0);
                    getRay(undistorted1, camMat1, camTransMat1, 
                            origin1, dir1);

                    float midPoint[4];

                    auto dist = getMidPoint(
                            origin0, dir0, origin1, dir1,
                            midPoint);
                    avgPoint[0] += midPoint[0];
                    avgPoint[1] += midPoint[1];
                    avgPoint[2] += midPoint[2];
                    avgPoint[3] += midPoint[3];
                    ptCount++;
                    if (dist < minDist)
                    {
                        minDist = dist;
                        memcpy (minMidPoint, midPoint, sizeof(float)*4);
                    }
                }
            //if (minDist < 0.3)
            avgPoint[0] /= (float)ptCount;
            avgPoint[1] /= (float)ptCount;
            avgPoint[2] /= (float)ptCount;
            avgPoint[3] = 1.0;
            memcpy ( &pointCloud[4*idx], avgPoint, sizeof(float)*4);
            //else
                //memset( &pointCloud[4*idx], 0, sizeof(float)*4);
        }
        idx += stride;
    }
}

} // namespace Kernel
} // namespace SLS
